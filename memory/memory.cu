#include <stdio.h>        // printf
#include <hip/hip_runtime.h> // cudaError_t


int main() {
    int a = 1;
    int* dev_a; // pointer to the device variable
    int b = 0;
    printf("Before sending value to GPU : a = %d, b = %d\n", a,b);

    // Alocate memory on GPU
    hipMalloc(&dev_a, sizeof(int));

    // Copy data on GPU
    hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);

    printf("After sending value to GPU : a = %d, b = %d\n", a,b);

    // Copy data from GPU
    hipMemcpy(&b, dev_a, sizeof(int),hipMemcpyDeviceToHost);
    
    printf("After retrieving value from GPU : a = %d, b = %d\n", a,b);

    hipFree(dev_a);


    int cpu_value = 1; // Value initialized on the host
    // Trying to copy data in a non allocated memory !
    hipError_t err = hipMemcpy(&cpu_value, &a, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\n----\nhipMemcpy error: %s\n", hipGetErrorString(err));
        return 1;
    }


    return 0;
}
