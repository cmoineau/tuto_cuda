#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_SAMPLES_PER_THREAD 10000
#define THREADS_PER_BLOCK 256

__global__ void monte_carlo_pi(int* d_num_points_inside_circle, int* d_num_points_total, hiprandState* d_state) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(clock64(), index, 0, &d_state[index]);

    int num_points_inside_circle = 0;
    for (int i = 0; i < NUM_SAMPLES_PER_THREAD; i++) {
        float x = hiprand_uniform(&d_state[index]);
        float y = hiprand_uniform(&d_state[index]);
        // Checkif random point is in the circle
        if (x * x + y * y <= 1.0f) {
            num_points_inside_circle++;
        }
    }
    // Use of atomicAdd to avoid race condition
    atomicAdd(d_num_points_inside_circle, num_points_inside_circle);
    atomicAdd(d_num_points_total, NUM_SAMPLES_PER_THREAD);
}

__global__ void setup_kernel(hiprandState* d_state, unsigned long long seed) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, tid, 0, &d_state[tid]);
}

int main() {
    int num_blocks = 1;
    int num_threads = THREADS_PER_BLOCK;
    int num_samples = NUM_SAMPLES_PER_THREAD * num_threads * num_blocks;

    // Allocate device memory for the random state and results
    hiprandState* d_state;
    hipMalloc(&d_state, sizeof(hiprandState) * num_threads * num_blocks);
    int* d_num_points_inside_circle;
    hipMalloc(&d_num_points_inside_circle, sizeof(int));
    int* d_num_points_total;
    hipMalloc(&d_num_points_total, sizeof(int));

    // Initialize the random state for each thread
    setup_kernel<<<num_blocks, num_threads>>>(d_state, time(NULL));

    // Launch the kernel
    monte_carlo_pi<<<num_blocks, num_threads>>>(d_num_points_inside_circle, d_num_points_total, d_state);

    // Copy the results back to the host
    int num_points_inside_circle, num_points_total;
    hipMemcpy(&num_points_inside_circle, d_num_points_inside_circle, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&num_points_total, d_num_points_total, sizeof(int), hipMemcpyDeviceToHost);

    // Compute the value of pi
    float pi = 4.0f * num_points_inside_circle / (float)num_points_total;

    // Print the result
    printf("Estimated value of pi: %f\n", pi);

    // Free the device memory
    hipFree(d_num_points_inside_circle);
    hipFree(d_num_points_total);
    hipFree(d_state);

    return 0;
}
