
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_kernel(int *a, int *b, int* result) {
    *result = *a + *b;
}

int main() {
    int a = 3;
    int b = 5;
    int c;
    int *dev_a, *dev_b, *dev_c;

    // Allocate space for device copied of a, b, c
    hipMalloc((void**) &dev_a, sizeof(int));
    hipMalloc((void**) &dev_b, sizeof(int));
    hipMalloc((void**) &dev_c, sizeof(int));
    

    hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice);


    add_kernel<<<1, 1>>>(dev_a, dev_b, dev_c);
    
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d + %d = %d\n", a, b, c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
