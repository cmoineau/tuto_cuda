#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_kernel(int a, int b, int* result) {
    *result = a + b;
}

int main() {
    int a = 3;
    int b = 5;
    int c;
    int* dev_c;
    
    hipError_t err = hipMalloc(&dev_c, sizeof(int));
    if (err != hipSuccess) {
        printf("hipMalloc error: %s\n", hipGetErrorString(err));
        return 1;
    }

    add_kernel<<<1, 1>>>(a, b, dev_c);
    err = hipDeviceSynchronize(); // Wait for kernel to finish
    if (err != hipSuccess) {
        printf("hipDeviceSynchronize error: %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("hipMemcpy error: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("%d + %d = %d\n", a, b, c);

    hipFree(dev_c);

    return 0;
}
