#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_SAMPLES_PER_THREAD 10000
#define THREADS_PER_BLOCK 256

__global__ void monte_carlo_pi(int* d_num_points_inside_circle, hiprandState* d_state) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(clock64(), index, 0, &d_state[index]);

    int num_points_inside_circle = 0;
    for (int i = 0; i < NUM_SAMPLES_PER_THREAD; i++) {
        float x = hiprand_uniform(&d_state[index]);
        float y = hiprand_uniform(&d_state[index]);
        // Checkif random point is in the circle
        if (x * x + y * y <= 1.0f) {
            num_points_inside_circle++;
        }
    }
    // Use of atomicAdd to avoid race condition
    atomicAdd(d_num_points_inside_circle, num_points_inside_circle);
}

__global__ void setup_kernel(hiprandState* d_state, unsigned long long seed) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, tid, 0, &d_state[tid]);
}

int main() {
    int num_streams = 4;
    int block_size = THREADS_PER_BLOCK;
    int num_blocks = 10;

    int* d_result[num_streams];
    int h_result[num_streams];
    hipStream_t stream[num_streams];
    hiprandState* d_state[num_streams];
    

    for (int i = 0; i < num_streams; i++) {
        hipMalloc(&d_state[i], sizeof(hiprandState) * block_size * num_blocks);
        hipStreamCreate(&stream[i]);
        hipMalloc(&d_result[i], sizeof(int));
        hipMemset(d_result[i], 0, sizeof(int));

        setup_kernel<<<num_blocks, block_size, 0, stream[i]>>>(d_state[i], time(NULL));
        monte_carlo_pi<<<num_blocks, block_size, 0, stream[i]>>>(d_result[i], d_state[i]);
    }

    for (int i = 0; i < num_streams; i++) {
        hipMemcpyAsync(&h_result[i], d_result[i], sizeof(float), hipMemcpyDeviceToHost, stream[i]);
        hipStreamSynchronize(stream[i]);
    }

    int nb_in_circle = 0;
    for (int i = 0; i < num_streams; i++) {
        nb_in_circle += h_result[i];
        printf("Steam %d : %f\n", i, 4.0f * h_result[i] / (num_blocks *block_size* NUM_SAMPLES_PER_THREAD));
        hipFree(d_result[i]);
        hipStreamDestroy(stream[i]);
    }

    float pi_estimate = 4.0f * nb_in_circle / (num_streams * num_blocks *block_size* NUM_SAMPLES_PER_THREAD);

    std::cout << "Pi estimate: " << pi_estimate << std::endl;

    return 0;
}
